#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <inttypes.h>
#include "skew_kernel.cuh"

//the first round construction
__global__ void get_sample_triplet(int *d_sa, char *d_buffer, int *d_sample, int mod31, int mod32, int size)
{
	int tid = TID;

	if(tid >= mod31)
		return;
	
	
	d_sa[tid] = tid; //3*tid+1;

	int pos = tid*3+1;
	int val1 = (pos<size)? d_buffer[pos] : 0;
	int val2 = (pos+1<size)? d_buffer[pos+1] : 0;
	int val3 = (pos+2<size)? d_buffer[pos+2] : 0;

	d_sample[tid] = (((val1&0x00ff) << 16) | ((val2&0x00ff) << 8) | (val3&0x00ff));

	if(tid < mod32)
	{
		d_sa[tid + mod31] = tid + mod31; //3*tid+2;

		pos = tid*3+2;
		val1 = (pos<size)? d_buffer[pos] : 0;
		val2 = (pos+1<size)? d_buffer[pos+1] : 0;
		val3 = (pos+2<size)? d_buffer[pos+2] : 0;

		d_sample[tid+mod31] = (((val1&0x00ff) << 16) | ((val2&0x00ff) << 8) | (val3&0x00ff));
	}
	
	return; 
}

//for the following round recursive construction
__global__ void get_sample_triplet_value1(int *d_intchar, uint64_t *d_sample, int *d_sa12, int mod31, int mod32, int size)
{
	int tid = TID;

	if(tid >= mod31)
		return;
	
	int pos = tid*3+1;
	//uint64 val1 = (pos<size)? d_intchar[pos] : 0;
	uint64_t val2 = (pos+1<size)? d_intchar[pos+1] : 0;
	uint64_t val3 = (pos+2<size)? d_intchar[pos+2] : 0;

	d_sample[tid] = ((val2&0xffffffff) << 32) | (val3&0xffffffff);
	d_sa12[tid]  = tid;//(tid*3+1);

	if(tid < mod32)
	{
		pos = tid*3+2;
		//val1 = (pos<size)? d_intchar[pos] : 0;
		val2 = (pos+1<size)?d_intchar[pos+1] : 0;
		val3 = (pos+2<size)? d_intchar[pos+2] : 0;

		d_sample[tid+mod31] = ((val2&0xffffffff) << 32) | (val3&0xffffffff);
		d_sa12 [tid+mod31] = /*(val1 << 32) | */tid+mod31; //(tid*3+2);
	}

	return;
}

//use the mid bit of d_value as the mark
__global__ void mark12(uint64_t *d_sample12, uint64_t *d_value, int mod31, int mod32)
{
	int tid = TID;

	if(tid >= mod31)
		return;

	uint64_t value = d_value[tid];

	if(tid == 0 || d_sample12[tid] != d_sample12[tid-1])
	{
		value = value | 0x80000000;
		d_value[tid] = value;
	}

	if(tid < mod32)
	{
		value = d_value[tid+mod31];

		if(d_sample12[tid+mod31] != d_sample12[tid+mod31-1])
		{
			value = value | 0x80000000;
			d_value[tid+mod31] = value;
		}
	}
}

//for the following round recursive construction
//get_sample_triplet_value2<<<h_dimGrid, h_dimBlock>>>(d_sa12, d_sample3, d_intchar, mod31, mod32);

__global__ void get_sample_triplet_value2(int *d_sa12, int *d_sample3, int *d_intchar, int mod31, int mod32)
{
	int tid = TID;

	if(tid >= mod31)
		return;

	int sa = d_sa12[tid];
	if(sa < mod31)
		sa = sa*3+1;
	else
		sa = (sa-mod31)*3+2;

	d_sample3[tid] = d_intchar[sa];

	if(tid < mod32)
	{
		sa = d_sa12[tid+mod31];
		if(sa < mod31)
			sa = sa*3+1;
		else
			sa = (sa-mod31)*3+2;

		d_sample3[tid+mod31] = d_intchar[sa];
	}

	return;
}


//use the mid bit of d_value as the mark
//mark3<<<h_dimGrid, h_dimBlock>>>(d_sample3, d_sa12, d_isa1, mod31, mod32);
__global__ void mark3(int *d_sample3, int *d_sa12, int *d_isa1, int *d_intchar, int mod31, int mod32, int size)
{
	int tid = TID;

	if(tid >= mod31)
		return;
	
	int mark = 0;
	int pos1, pos2, val11, val12, val21, val22;

	if(tid == 0)
		mark = 1;
	else if(d_sample3[tid] != d_sample3[tid-1])
		mark = 1;
	else
	{
		pos1 = d_sa12[tid];
		pos2 = d_sa12[tid-1];

		if(pos1 < mod31)
			pos1 = pos1*3+1;
		else
			pos1 = (pos1-mod31)*3+2;

		if(pos2 < mod31)
			pos2 = pos2*3+1;
		else
			pos2 = (pos2-mod31)*3+2;

		val11 = (pos1+1<size)?d_intchar[pos1+1] : 0;
		val12 = (pos1+2<size)?d_intchar[pos1+2] : 0;

		val21 = (pos2+1<size)?d_intchar[pos2+1] : 0;
		val22 = (pos2+2<size)?d_intchar[pos2+2] : 0;

		if(val11 != val21 || val12 != val22)
			mark = 1;
	}

	d_isa1[tid] = mark;

	if(tid < mod32)
	{
		mark = 0;

		if(d_sample3[tid+mod31] != d_sample3[tid+mod31-1])
			mark = 1;
		else
		{
			pos1 = d_sa12[tid+mod31];
			pos2 = d_sa12[tid+mod31-1];

			if(pos1 < mod31)
				pos1 = pos1*3+1;
			else
				pos1 = (pos1-mod31)*3+2;

			if(pos2 < mod31)
				pos2 = pos2*3+1;
			else
				pos2 = (pos2-mod31)*3+2;

			val11 = (pos1+1<size)?d_intchar[pos1+1] : 0;
			val12 = (pos1+2<size)?d_intchar[pos1+2] : 0;

			val21 = (pos2+1<size)?d_intchar[pos2+1] : 0;
			val22 = (pos2+2<size)?d_intchar[pos2+2] : 0;

			if(val11 != val21 || val12 != val22)
				mark = 1;
		}
		d_isa1[tid+mod31] = mark;
	}
}

__global__ void neighbour_comparison_kernel1(int *d_mark, int *d_sample, int sample_size)
{
	//times 4
	int tid = (TID << 2);

	if (tid >= sample_size)
		return;

	int4* d_mark_ptr = (int4*)(d_mark+tid);
	int4  key_data = *((int4*)(d_sample+tid));

	int4 out;
	
	if(tid == 0)
		out.x = 1;
	
	if (key_data.x == key_data.y)
		out.y = 0;
	else
		out.y = 1;

	if (key_data.y == key_data.z)
		out.z = 0;
	else
		out.z = 1;

	if (key_data.z == key_data.w)
		out.w = 0;
	else
		out.w = 1;

	*d_mark_ptr = out;
}


__global__ void neighbour_comparison_kernel2(int *d_mark, int *d_sample, int sample_size)
{

	int tid = ((TID)+1) << 2;
	
	if (tid >= sample_size)
		return;

	if (d_sample[tid] == d_sample[tid-1])
		d_mark[tid] = 0;
	else
		d_mark[tid] = 1;
}


__global__ void scatter_global_rank(int *d_sa12, int *d_globalRank, int sample_size, int string_size)
{
	int tid = TID;
	if(tid >= sample_size)
		return;

	int sa = d_sa12[tid];

	d_globalRank[sa] = tid+1;
}

__global__ void scatter_for_recursion(int *d_isa_in, int *d_isa_out, int *d_sa12, int mod31, int sample_size)
{
	int tid = TID;
	if(tid >= sample_size)
		return;

	int rank = d_isa_in[tid];
	int sa12 = d_sa12[tid];

	d_isa_out[sa12] = rank;
}

__global__ void transform_local2global_sa(int *d_sa, int mod31, int sample_size)
{
	int tid = TID;
	if(tid >= sample_size)
		return;

	int sa = d_sa[tid];
	if(sa < mod31)
		d_sa[tid] = sa*3+1;
	else
		d_sa[tid] = (sa-mod31)*3+2;
}


template<typename T>
__global__ void bitonic_sort_step(int *d_sa, int *d_global_rank, T *d_intchar, int sample_len, int j)
{
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	int ixj = i^j;
 
}
