#include <iostream>
#include <fstream>
#include <sstream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <algorithm>
#include <vector>

#include <thrust/count.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/unique.h>
#include <thrust/merge.h>
#include <thrust/set_operations.h>

#include <moderngpu.cuh>
#include <kernels/mergesort.cuh>
#include <kernels/segmentedsort.cuh>

#include "skew_kernel.cuh"

mgpu::ContextPtr context;

using namespace std;
using namespace mgpu;

float	radixsort = 0.0;
float	mergesort = 0.0;
float  	ranktriplet = 0.0;
float	others = 0.0;
float	computesa = 0.0;


void cudaCheckError(int line)
{
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	if(err != hipSuccess)
		printf("Last cuda error is %d at %d\n", err, line);
}

void checkMemoryUsage(int line)
{
	size_t freed;
	size_t total;
	hipMemGetInfo(&freed, &total);
	printf("Line %d: free memory is %zd, and total is %zd\n", line, freed, total);
}


template<typename T1, typename T2>
void sort(T1 *d_key, T2 *d_value, int size)
{
	thrust::device_ptr<T1> d_key_ptr  = thrust::device_pointer_cast(d_key);
	thrust::device_ptr<T2> d_val_ptr = thrust::device_pointer_cast(d_value);
	thrust::sort_by_key(d_key_ptr, d_key_ptr+size, d_val_ptr);
}

int prefix_sum(int *d_input, int *d_output, int size)
{
	int sum = 0;

	//int32 first_rank = 1;
	//mem_host2device(&first_rank, d_input, sizeof(int32));

	cudaCheckError(__LINE__);

	thrust::device_ptr<int> d_input_ptr = thrust::device_pointer_cast(d_input);
	thrust::device_ptr<int> d_output_ptr = thrust::device_pointer_cast(d_output);

	thrust::inclusive_scan(d_input_ptr, d_input_ptr+size, d_output_ptr);

	hipMemcpy(&sum, d_output+size-1, sizeof(int), hipMemcpyDeviceToHost);

	return sum;
}

void recursiveSort(int *d_intchar, int *d_sa, int size)
{

    	float time;
    	hipEvent_t start;
    	hipEvent_t stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);

	//construct sample string
	//exclude the last \0

	int mod30 = size/3 + (size%3!=0);
	int mod31 = size/3 + (size%3==2);
	int mod32 = size/3;

	int sample_len = mod31 + mod32;

	uint64_t *d_sample12;
	int   *d_sa12, *d_sa12_t;

	//construct and sort the first part key 
	//(the first two int of the triplet, and the second int is in the value part with the sa array)
	hipMalloc((void**)&d_sample12, (sample_len+1)*sizeof(uint64_t));
	hipMalloc((void**)&d_sa12,  	(sample_len+1)*sizeof(int));

	time = 0.0;
	hipEventRecord(start, 0);

	dim3 h_dimBlock(BLOCK_SIZE,1,1);
	dim3 h_dimGrid(1,1,1);
	int numBlocks = CEIL(mod31, h_dimBlock.x);
	THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);

	get_sample_triplet_value1<<<h_dimGrid, h_dimBlock>>>(d_intchar, d_sample12, d_sa12, mod31, mod32, size);

	thrust::device_ptr<uint64_t> d_key_ptr  = thrust::device_pointer_cast(d_sample12);
	thrust::device_ptr<int>   d_sa12_ptr = thrust::device_pointer_cast(d_sa12);
	thrust::sort_by_key(d_key_ptr, d_key_ptr+sample_len, d_sa12_ptr);

	int *d_sample3 = (int*)d_sample12;
	int *d_isa1    = d_sample3 + sample_len;

	//construct and sort the second part key
	get_sample_triplet_value2<<<h_dimGrid, h_dimBlock>>>(d_sa12, d_sample3, d_intchar, mod31, mod32);

	thrust::device_ptr<int> d_sample_ptr = thrust::device_pointer_cast(d_sample3);
	thrust::stable_sort_by_key(d_sample_ptr, d_sample_ptr+sample_len, d_sa12_ptr);

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
   	hipEventElapsedTime(&time, start, stop);
	radixsort += time;

	//////////////////////compute ISA

	time = 0.0;
	hipEventRecord(start, 0);

	mark3<<<h_dimGrid, h_dimBlock>>>(d_sample3, d_sa12, d_isa1, d_intchar, mod31, mod32, size);
	int *d_isa2 = d_sample3;
	//prefix sum
	int num_unique = prefix_sum(d_isa1, d_isa2, sample_len);

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
   	hipEventElapsedTime(&time, start, stop);
	ranktriplet += time;

	//////////////////////////////////////////////////////
	printf("num_unique2 is %d, %d\n", num_unique, sample_len);

	if(num_unique != sample_len)
	{	
		time = 0.0;
		hipEventRecord(start, 0);
	
		//scatter to compute isa
		h_dimGrid.x = h_dimGrid.y = 1;
		numBlocks = CEIL(sample_len, h_dimBlock.x);
		THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);
		scatter_for_recursion<<<h_dimGrid, h_dimBlock>>>(d_isa2, d_isa1, d_sa12, mod31, sample_len);

		hipMemset(d_isa1+sample_len, 0, sizeof(int));

   		hipEventRecord(stop, 0);
    		hipEventSynchronize(stop);
   		hipEventElapsedTime(&time, start, stop);
		others += time;

		//recursive sort
		recursiveSort(d_isa1, d_sa12, sample_len+1);
		d_sa12_t = d_sa12+1;
	}
	else
		d_sa12_t = d_sa12;

	hipFree(d_sample12);

	time = 0.0;
	hipEventRecord(start, 0);

	h_dimGrid.x = h_dimGrid.y = 1;
	numBlocks = CEIL(sample_len, h_dimBlock.x);
	THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);
	transform_local2global_sa<<<h_dimGrid, h_dimBlock>>>(d_sa12_t, mod31, sample_len);

	int *d_global_rank;
	hipMalloc((void**)&d_global_rank, (size+2)*sizeof(int));
	hipMemset(d_global_rank, -1, size*sizeof(int));
	hipMemset(d_global_rank+size, 0, 2*sizeof(int));

	h_dimGrid.x = h_dimGrid.y = 1;
	numBlocks = CEIL(sample_len, h_dimBlock.x);
	THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);

	//scatter d_sa12 to get global rank for pos mod31 and mod32, as well as size and size+1
	scatter_global_rank<<<h_dimGrid, h_dimBlock>>>(d_sa12_t, d_global_rank, sample_len, size);

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
 	hipEventElapsedTime(&time, start, stop);
	computesa += time;

	////////////////////////////////////radix sort sa0
	//mod30 = size - sample_len;
	uint64_t *d_key0;
	int *d_sa0;
	hipMalloc((void**)&d_key0, mod30*sizeof(uint64_t));
	hipMalloc((void**)&d_sa0, mod30*sizeof(int));	

	time = 0.0;
	hipEventRecord(start, 0);

	h_dimGrid.x = h_dimGrid.y = 1;
	numBlocks = CEIL(mod30, h_dimBlock.x);
	THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);
	get_s0_pair<int, false><<<h_dimGrid, h_dimBlock>>>(d_key0, d_sa0, d_intchar, d_global_rank, mod30, size);

	thrust::device_ptr<uint64_t> d_key0_ptr = thrust::device_pointer_cast(d_key0);
	thrust::device_ptr<int> d_sa0_ptr = thrust::device_pointer_cast(d_sa0);
	thrust::sort_by_key(d_key0_ptr, d_key0_ptr+mod30, d_sa0_ptr);

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
   	hipEventElapsedTime(&time, start, stop);
	radixsort += time;

	///////////////////////////merge sort	
	time = 0.0;
	hipEventRecord(start, 0);

	//merge s0 and s12
	thrust::device_ptr<int> d_global_sa_ptr1 = thrust::device_pointer_cast(d_sa);
	thrust::device_ptr<int> d_sa0_ptr1 = thrust::device_pointer_cast(d_sa0);
	thrust::device_ptr<int> d_sa12_ptr1 = thrust::device_pointer_cast(d_sa12_t);
	
	//mgpu::MergeKeys(d_sa0, mod30, d_sa12_t, sample_len, d_sa, merge_comp_int(d_intchar, d_global_rank, sample_len, size), *context);

	thrust::merge(d_sa0_ptr1, d_sa0_ptr1+mod30, d_sa12_ptr1, d_sa12_ptr1+sample_len, d_global_sa_ptr1, merge_comp_int(d_intchar, d_global_rank, sample_len, size));

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
   	hipEventElapsedTime(&time, start, stop);
	mergesort += time;

	hipFree(d_key0);
	hipFree(d_sa0);
	hipFree(d_global_rank);
	hipFree(d_sa12);

	hipEventDestroy(start);
	hipEventDestroy(stop);

}

void computeSA(char *d_buffer, int *d_global_sa, char *h_buffer, int size)
{
    	float time;
    	hipEvent_t start;
    	hipEvent_t stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);

	//construct sample string
	//size-1 or not
	int mod30 = (size)/3 + ((size)%3!=0);
	int mod31 = (size)/3 + ((size)%3==2);
	int mod32 = (size)/3;

	int sample_len = mod31 + mod32;

	printf("num elements mod3 is 0,1,2 is %d, %d, %d\n", mod30, mod31, mod32);

	int *d_sample, *d_sa12, *d_isa1, *d_isa2, *d_sa12_t;
	hipMalloc((void**)&d_sample, (sample_len+3)*sizeof(int));
	hipMalloc((void**)&d_sa12,   (sample_len+3)*sizeof(int));
	hipMalloc((void**)&d_isa1,   (sample_len+3)*sizeof(int));
	d_isa2 = d_sample;
	//hipMalloc((void**)&d_isa2,   (sample_len+3)*sizeof(int));

	dim3 h_dimBlock(BLOCK_SIZE,1,1);
	dim3 h_dimGrid(1,1,1);
	int numBlocks = CEIL(mod31, h_dimBlock.x);
	THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);

	time = 0.0;
	hipEventRecord(start, 0);

	get_sample_triplet<<<h_dimGrid, h_dimBlock>>>(d_sa12, d_buffer, d_sample, mod31, mod32, size);

	//sort the triplets
	thrust::device_ptr<int> d_sample_ptr = thrust::device_pointer_cast(d_sample);
	thrust::device_ptr<int> d_sa12_ptr = thrust::device_pointer_cast(d_sa12);
	thrust::sort_by_key(d_sample_ptr, d_sample_ptr+sample_len, d_sa12_ptr);

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
   	hipEventElapsedTime(&time, start, stop);
	radixsort += time;

	//////////////////////////compute ISA

	time = 0.0;
	hipEventRecord(start, 0);

	int last_rank[] = {0xffffffff, 0, 0xffffffff};
	hipMemcpy(d_isa1+sample_len, last_rank, sizeof(int)*3, hipMemcpyHostToDevice);

	h_dimGrid.x = h_dimGrid.y = 1;
	numBlocks = CEIL(CEIL(sample_len, 4), h_dimBlock.x);
	THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);

	//mark the start position of each segment to 1
	neighbour_comparison_kernel1<<<h_dimGrid, h_dimBlock>>>(d_isa1, d_sample, sample_len);
	neighbour_comparison_kernel2<<<h_dimGrid, h_dimBlock>>>(d_isa1, d_sample, sample_len);

	int num_unique = prefix_sum(d_isa1, d_isa2, sample_len);

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
   	hipEventElapsedTime(&time, start, stop);
	ranktriplet += time;

	printf("num_unique is %d, %d\n", num_unique, sample_len);

	if(num_unique != sample_len)
	{
		time = 0.0;
		hipEventRecord(start, 0);

		//scatter to compute isa
		h_dimGrid.x = h_dimGrid.y = 1;
		numBlocks = CEIL(sample_len, h_dimBlock.x);
		THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);
		scatter_for_recursion<<<h_dimGrid, h_dimBlock>>>(d_isa2, d_isa1, d_sa12, mod31, sample_len);
		hipMemset(d_isa1+sample_len, 0, sizeof(int));

   		hipEventRecord(stop, 0);
    		hipEventSynchronize(stop);
   		hipEventElapsedTime(&time, start, stop);
		others += time;

		//recursive sort
		recursiveSort(d_isa1, d_sa12, sample_len+1);
		d_sa12_t = d_sa12+1;
	}
	else
		d_sa12_t = d_sa12;

	time = 0.0;
	hipEventRecord(start, 0);

	h_dimGrid.x = h_dimGrid.y = 1;
	numBlocks = CEIL(sample_len, h_dimBlock.x);
	THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);
	transform_local2global_sa<<<h_dimGrid, h_dimBlock>>>(d_sa12_t, mod31, sample_len);
	
	cudaCheckError(__LINE__);

	int *d_global_rank;
	hipMalloc((void**)&d_global_rank, (size+2)*sizeof(int));
	hipMemset(d_global_rank, 0, (size+2)*sizeof(int));

	h_dimGrid.x = h_dimGrid.y = 1;
	numBlocks = CEIL(sample_len, h_dimBlock.x);
	THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);

	//scatter d_sa12 to get global rank for pos mod31 and mod32, as well as size and size+1
	scatter_global_rank<<<h_dimGrid, h_dimBlock>>>(d_sa12_t, d_global_rank, sample_len, size);

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
   	hipEventElapsedTime(&time, start, stop);
	computesa += time;


	//radix sort s0
	mod30 = size - sample_len;
	uint64_t *d_key0;
	int *d_sa0;
	hipMalloc((void**)&d_key0, mod30*sizeof(uint64_t));
	hipMalloc((void**)&d_sa0, mod30*sizeof(int));
	
	time = 0.0;
	hipEventRecord(start, 0);

	h_dimGrid.x = h_dimGrid.y = 1;
	numBlocks = CEIL(mod30, h_dimBlock.x);
	THREAD_CONF(h_dimGrid, h_dimBlock, numBlocks, h_dimBlock.x);
	get_s0_pair<char, true><<<h_dimGrid, h_dimBlock>>>(d_key0, d_sa0, d_buffer, d_global_rank, mod30, size);

	thrust::device_ptr<uint64_t> d_key0_ptr = thrust::device_pointer_cast(d_key0);
	thrust::device_ptr<int> d_sa0_ptr = thrust::device_pointer_cast(d_sa0);
	thrust::sort_by_key(d_key0_ptr, d_key0_ptr+mod30, d_sa0_ptr);

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
   	hipEventElapsedTime(&time, start, stop);
	radixsort += time;

	/////////////////////////////merge sa0 and sa12
	time = 0.0;
	hipEventRecord(start, 0);

	thrust::device_ptr<int> d_global_sa_ptr = thrust::device_pointer_cast(d_global_sa);
	thrust::device_ptr<int> d_sa12_ptr1 = thrust::device_pointer_cast(d_sa12_t);


	//mgpu::MergeKeys(d_sa0, mod30, d_sa12_t, sample_len, d_global_sa, merge_comp_char(d_buffer, d_global_rank, sample_len, size), *context);

	thrust::merge(d_sa0_ptr, d_sa0_ptr+mod30, d_sa12_ptr1, d_sa12_ptr1+sample_len, d_global_sa_ptr, merge_comp_char(d_buffer, d_global_rank, sample_len, size));

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
   	hipEventElapsedTime(&time, start, stop);
	mergesort += time;

	hipFree(d_sa12);
	hipFree(d_sample);
	hipFree(d_isa1);
	hipFree(d_global_rank);
	hipFree(d_key0);
	hipFree(d_sa0);

	hipEventDestroy(start);
	hipEventDestroy(stop);
}


int main(int argc, char** argv)
{
	if(argc < 2)
	{
		printf("file name!\n");
		exit(-1);
	}
	
	////////////////
	FILE * pFile;
  	long size;
	size_t result;

 	pFile = fopen (argv[1],"r");
	if (pFile==NULL) { perror ("Error opening file\n"); exit(1); }

    	fseek (pFile, 0, SEEK_END);
    	size=ftell(pFile);
	rewind (pFile);	
    	printf ("file size is: %ld bytes.\n",size);

	char *h_buffer = (char*)malloc((size+4)*sizeof(char));
	if (h_buffer == NULL) {fputs ("Memory error",stderr); exit (2);}
	
  	// copy the file into the buffer:
  	result = fread (h_buffer,1, size, pFile);
  	if (result != size) {fputs ("Reading error",stderr); exit (3);}

	h_buffer[size] = h_buffer[size+1] = h_buffer[size+2] = h_buffer[size+3] = 0;

	printf("last char is %d\n", h_buffer[size-1]);

	/*
	if(h_buffer[size-1] == 10)
	{	
		h_buffer[size-1]=0;
	}*/

	if(h_buffer[size-1] != 0)
	{	
		size+=1;
	}
	
	printf("string size is %ld\n", size);
	fclose (pFile);

	//context = CreateCudaDevice(0);

	int *h_sa = (int*)malloc(size*sizeof(int));

	char *d_buffer;
	hipMalloc((void**)&d_buffer,  	(size+3)*sizeof(char));
	hipMemcpy(d_buffer, h_buffer, (size+3)*sizeof(char), hipMemcpyHostToDevice);

	int *d_sa;
	hipMalloc((void**)&d_sa, size*sizeof(int));

    	float time;
    	hipEvent_t start;
    	hipEvent_t stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	computeSA(d_buffer, d_sa, h_buffer, size);

   	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
    	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	hipMemcpy(h_sa,  d_sa,  size*sizeof(int), hipMemcpyDeviceToHost);


	printf("radix sort time is %f\n", radixsort);
	printf("merge sort time is %f\n", mergesort);
	printf("rank triplet time is %f\n", ranktriplet);
	printf("compute SA time is %f\n", computesa);
	printf("others time is %f\n", others);
	printf("skew suffix sort time is %f, %f\n", time, radixsort+mergesort+ranktriplet+computesa+others);


	check_h_order_correctness(h_sa, h_buffer, size, size, size);

	hipFree(d_buffer);
	hipFree(d_sa);
	free(h_buffer);
	free(h_sa);
}
